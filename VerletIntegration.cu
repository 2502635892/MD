#include "hip/hip_runtime.h"
__global__ void UpdateAtomsInfo_kernel_1(AtomInfo *d_atoms, int N, double dt)
{
  int id = threadIdx.x + blockIdx.x * blockDim.x; //this atom's id

  //the simulation area Box size
  double Box_Lx = L_x * a_x;
  double Box_Ly = L_y * a_y;
  double Box_Lz = L_z * a_z;  

  dt = dt / TIME_UNIT_CONVERSION;

  if (id < N)
  {
    d_atoms[id].velocity[0] = d_atoms[id].velocity[0] + d_atoms[id].force[0] / d_atoms[id].mass * dt * 0.5;
    d_atoms[id].velocity[1] = d_atoms[id].velocity[1] + d_atoms[id].force[1] / d_atoms[id].mass * dt * 0.5;
    d_atoms[id].velocity[2] = d_atoms[id].velocity[2] + d_atoms[id].force[2] / d_atoms[id].mass * dt * 0.5;

    d_atoms[id].position[0] = d_atoms[id].position[0] + d_atoms[id].velocity[0] * dt;
    d_atoms[id].position[1] = d_atoms[id].position[1] + d_atoms[id].velocity[1] * dt;
    d_atoms[id].position[2] = d_atoms[id].position[2] + d_atoms[id].velocity[2] * dt; 
    
    // pbc operation
    d_atoms[id].position[0] = d_atoms[id].position[0] - Box_Lx * floor(d_atoms[id].position[0] / Box_Lx);
    d_atoms[id].position[1] = d_atoms[id].position[1] - Box_Ly * floor(d_atoms[id].position[1] / Box_Ly);
    d_atoms[id].position[2] = d_atoms[id].position[2] - Box_Lz * floor(d_atoms[id].position[2] / Box_Lz); //PBC
  }

}




void Update_AtomsInfo_kernel_1(AtomInfo **d_atoms , int* h_ParticleCount , double dt,int deviceCount)
{
    for(int i=0;i<deviceCount;i++)
  {
    int dimgrid;
    dim3 dimBlock = 128;
    int N =  h_ParticleCount[i];   // subdomain has atom number
    if (N % 128 == 0)
      dimgrid = N / 128;
    else
      dimgrid = N / 128 + 1;

    dim3 dimGrid = dimgrid;

    hipSetDevice(i);
    UpdateAtomsInfo_kernel_1<<<dimGrid,dimBlock>>>(d_atoms[i], N, dt);
    CUDA_CHECK_ERROR();

   }
    // 更新 执行同步
   All_devices_Synchronize(deviceCount);

}






__global__ void test_UpdateAtomsInfo_kernel_1(AtomInfo *d_atoms, int N ,TableInfo T_Info ,deProcess P , int sudDomainId)
{
  int id = threadIdx.x + blockIdx.x * blockDim.x;  //this atom's id

  if( id == 0 )
  {
      int cx,cy,cz;
      int subdomain_id;
      int ecape_count = 0 ;

      for(int i=0;i<N;i++)
    {
         if(i==0 || i==(N-1) )
        {
            printf("%dth: P: %f %f %f \t  V:%f %f %f \t F:%f %f %f\n",i,d_atoms[i].position[0],d_atoms[i].position[1],d_atoms[i].position[2],
            d_atoms[i].velocity[0],d_atoms[i].velocity[1],d_atoms[i].velocity[2], 
            d_atoms[i].force[0],d_atoms[i].force[1],d_atoms[i].force[2]);
        }

        cx = (int)(d_atoms[i].position[0] / T_Info.L_cell[0]);
        cy = (int)(d_atoms[i].position[1] / T_Info.L_cell[1]);
        cz = (int)(d_atoms[i].position[2] / T_Info.L_cell[2]);

        subdomain_id = d_cellToSubDomainId(cx , cy , cz , T_Info , P );

        if(subdomain_id != sudDomainId )
            ecape_count ++ ; 
     

    }

    printf("the %dth gpu check escape number is %d\n ",sudDomainId,ecape_count);

  }


}




void test_Update_AtomsInfo_kernel_1(AtomInfo **d_atoms , int* h_ParticleCount ,TableInfo T_Info ,int deviceCount,deProcess P )
{
  
     for(int i=0;i<deviceCount;i++) 
    { 
      dim3 testBlock = 128;
      dim3 testGrid = 1;
      int N = h_ParticleCount[i];
      hipSetDevice(i);
      test_UpdateAtomsInfo_kernel_1<<<testGrid,testBlock>>>(d_atoms[i],  N , T_Info , P , i);
      CUDA_CHECK_ERROR();
    }

}






__global__ void UpdateAtomsInfo_kernel_2(AtomInfo *d_atoms, int N, double dt)
{
  int id = threadIdx.x + blockIdx.x * blockDim.x; //this atom's id

  dt = dt / TIME_UNIT_CONVERSION;

  if (id < N)
  {
    d_atoms[id].velocity[0] = d_atoms[id].velocity[0] + d_atoms[id].force[0] / d_atoms[id].mass * dt * 0.5;
    d_atoms[id].velocity[1] = d_atoms[id].velocity[1] + d_atoms[id].force[1] / d_atoms[id].mass * dt * 0.5;
    d_atoms[id].velocity[2] = d_atoms[id].velocity[2] + d_atoms[id].force[2] / d_atoms[id].mass * dt * 0.5;
  }
  
}

void Update_AtomsInfo_kernel_2(AtomInfo **d_atoms , int* h_ParticleCount ,double dt,int deviceCount)
{
    for(int i=0;i<deviceCount;i++)
  {
    int dimgrid;
    dim3 dimBlock = 128;
    int N =  h_ParticleCount[i];  // subdomain has atom number
    if (N % 128 == 0)
      dimgrid = N / 128;
    else
      dimgrid = N / 128 + 1;

    dim3 dimGrid = dimgrid;

    hipSetDevice(i);
    UpdateAtomsInfo_kernel_2<<<dimGrid,dimBlock>>>(d_atoms[i], N, dt);
    CUDA_CHECK_ERROR();

   }

   All_devices_Synchronize(deviceCount);

}








__global__ void Scale_Velocity_kernel(AtomInfo* d_atoms , int N , double double_ek , int Total_Particle )
{
   double scalar; 
   int id = threadIdx.x +  blockIdx.x * blockDim.x; // this atom's id  
   if( id < N )
   {
      scalar = sqrt( Termp * Dim * K_B * Total_Particle / double_ek); // scale coefficient 

      d_atoms[id].velocity[0] = d_atoms[id].velocity[0] * scalar;
      d_atoms[id].velocity[1] = d_atoms[id].velocity[1] * scalar;
      d_atoms[id].velocity[2] = d_atoms[id].velocity[2] * scalar;
   }
       
}



void Scale_Velocity(AtomInfo **d_atoms ,int* h_ParticleCount , double double_ek , int deviceCount , int Total_Particle)
{
    for(int i=0;i<deviceCount;i++)
  {
    int dimgrid;
    dim3 dimBlock = 128;
    int N =  h_ParticleCount[i];  // subdomain has atom number
    if (N % 128 == 0)
      dimgrid = N / 128;
    else
      dimgrid = N / 128 + 1;

    dim3 dimGrid = dimgrid;

    hipSetDevice(i);
    Scale_Velocity_kernel<<<dimGrid,dimBlock>>>(d_atoms[i] , N , double_ek ,Total_Particle);
    CUDA_CHECK_ERROR();

   }
   
   All_devices_Synchronize(deviceCount);

}




















