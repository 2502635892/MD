#include "hip/hip_runtime.h"
void d_LingerCount_Set_Zero(int **d_LingerCount,int deviceCount)
{
    for(int i=0;i<deviceCount;i++)
  {
    hipSetDevice(i);
    CUDA_CALL( hipMemset(d_LingerCount[i] , 0 , sizeof(int) ) ); //set ReceCount[i] to 0
  }

}


void d_Escape_SendCount_Set_Zero(int **d_Escape_SendCount , int deviceCount)
{
    for(int i=0;i<deviceCount;i++)
    {
      hipSetDevice(i);
      CUDA_CALL( hipMemset(d_Escape_SendCount[i] , 0 , sizeof(int)*deviceCount ) ); //set SendCount[i] to 0
    } 
   // hipMemset() 是同步调用函数
}





__global__ void Adjust_kernel(AtomInfo *d_atoms , int N , AtomInfo *d_Adjust ,int *d_LingerCount, AtomInfo *SendBuffer ,int *d_Escape_SendCount , int SegInterval , deProcess P , TableInfo T_Info, int subDomainId,int Pnum )
{
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  int cx,cy,cz;
  int J_subDomainId;
  int atomInf_offset;
  int send_offset;

  if(id < N)
  {
      cx = (int)(d_atoms[id].position[0] / T_Info.L_cell[0]);  //  get the atom cell coordinate
      cy = (int)(d_atoms[id].position[1] / T_Info.L_cell[1]);
      cz = (int)(d_atoms[id].position[2] / T_Info.L_cell[2]);

      J_subDomainId = d_cellToSubDomainId(cx , cy , cz , T_Info , P );
    
      if(J_subDomainId == subDomainId)              // not escape atom
      {
        atomInf_offset = atomicAdd(&(d_LingerCount[0]) , 1); // get the adjustment array index parallel
        d_Adjust[atomInf_offset] = d_atoms[id];      // insert into  d_Adjust array when it is not escape 
      }
      else
      {
           send_offset = atomicAdd(&d_Escape_SendCount[J_subDomainId],1);
           if( send_offset < SegInterval )
	      {
              SendBuffer[ SegInterval * J_subDomainId + send_offset ] = d_atoms[id];
	      }
	       else
	      {
	           printf("out of buffer\n");
	      }
      }
  }

}



// int* h_ParticleCount

void Split_Escape_And_linger(AtomInfo ** d_atoms ,int* h_ParticleCount , AtomInfo **d_Adjust, int **d_LingerCount ,AtomInfo **SendBuffer, int **d_Escape_SendCount ,int deviceCount,deProcess P,TableInfo T_Info ,int Pnum ,int SegInterval)
{
    for(int i=0;i<deviceCount;i++)
  { 
    int dimgrid;
    dim3 dimBlock = 128;
    int N =  h_ParticleCount[i];  // subdomain has atom number

    if (N % 128 == 0)
      dimgrid = N / 128;
    else
      dimgrid = N / 128 + 1;

    dim3 dimGrid = dimgrid;

    hipSetDevice(i);
    Adjust_kernel<<<dimGrid,dimBlock>>>(d_atoms[i] , N , d_Adjust[i] , d_LingerCount[i], SendBuffer[i] , d_Escape_SendCount[i] , SegInterval , P , T_Info, i, Pnum );

    CUDA_CHECK_ERROR();

   } //send the escape atom to sendbuffer and the linger atom to the d_Adjust array
 
   All_devices_Synchronize(deviceCount);

}



void d_Escape_SendCount_DeviceToHost(int **d_Escape_SendCount ,int ** h_Escape_SendCount , int deviceCount)
{
     for(int i=0;i<deviceCount;i++)
    {
      hipSetDevice(i);
      CUDA_CALL( hipMemcpy(h_Escape_SendCount[i] , d_Escape_SendCount[i] , sizeof(int) * deviceCount , hipMemcpyDeviceToHost) );
    }

}



void test_d_Escape_SendCount(int ** h_Escape_SendCount , int deviceCount)
{
    for(int i=0;i<deviceCount;i++)
    {
        printf("the %dth GPU escape particle number is : ",i);
         for(int j=0;j<deviceCount;j++)
        {
            printf("%d\t",h_Escape_SendCount[i][j]);
        }
        printf("\n");
    }

}



void d_Escape_ReceiveCount_From_SendCount(int ** d_Escape_ReceCount,int **d_Escape_SendCount , int deviceCount)
{
  for(int i=0;i<deviceCount;i++)  // ith gpu send
  { 
      for(int j=0;j<deviceCount;j++)
     {
        CUDA_CALL( hipMemcpyPeer( &(d_Escape_ReceCount[i][j]) , i , &(d_Escape_SendCount[j][i]) , j , sizeof(int) )  );
     } // 串行通信
  } 

}



void d_Escape_ReceCount_DeviceToHost(int ** h_Escape_ReceCount , int ** d_Escape_ReceCount , int deviceCount)
{
    for(int i=0;i<deviceCount;i++)
    {
      hipSetDevice(i);
      CUDA_CALL( hipMemcpy( h_Escape_ReceCount[i] , d_Escape_ReceCount[i] , sizeof(int) * deviceCount , hipMemcpyDeviceToHost ) );
    }
    
}



void LingerCount_DeviceToHost(int **h_LingerCount , int **d_LingerCount,int deviceCount)
{
  for(int i=0;i<deviceCount;i++)
  {
    hipSetDevice(i);
    CUDA_CALL( hipMemcpy( h_LingerCount[i] , d_LingerCount[i] , sizeof(int) , hipMemcpyDeviceToHost ) );
  }

}




void linger_To_AtomInfoArray(AtomInfo ** d_atoms , AtomInfo **d_Adjust , int **h_LingerCount ,int deviceCount)
{
    for(int i=0;i<deviceCount;i++)
   {
     hipSetDevice(i);
     CUDA_CALL( hipMemcpy( d_atoms[i] , d_Adjust[i] , sizeof(AtomInfo) * h_LingerCount[i][0] , hipMemcpyDeviceToDevice)  );
   }

}





__global__ void test_linger_To_AtomInfo_kernel(AtomInfo *d_atoms, AtomInfo *d_Adjust ,int *d_LingerCount ,int thisDevice)
{
  int id = threadIdx.x + blockIdx.x * blockDim.x;  //this atom's id

  if( id == 0 )
  {
      int count = 0 ;  
      for(int i=0;i<d_LingerCount[0];i++)
    {
        bool b1 = ( d_atoms[i].velocity[0] == d_Adjust[i].velocity[0] );
        bool b2 = ( d_atoms[i].velocity[1] == d_Adjust[i].velocity[1] );
        bool b3 = ( d_atoms[i].velocity[2] == d_Adjust[i].velocity[2] );

        if(b1&&(b2&&b3))
           count ++;
      
    }
     if(count==d_LingerCount[0])
         printf("\n\nthe %dth gpu check linger insert %d number partcle is successfully! \n\n ",thisDevice,count);
      else
        printf("\nlinger error on %dth GPU\n",thisDevice);

  }


}



void test_linger_To_AtomInfo(AtomInfo **d_atoms, AtomInfo **d_Adjust ,int **d_LingerCount , int deviceCount)
{
  for(int i=0;i<deviceCount;i++)
  {
    dim3 testBlock = 128;
    dim3 testGrid = 1;
    hipSetDevice(i);
    test_linger_To_AtomInfo_kernel<<<testGrid,testBlock>>>(d_atoms[i], d_Adjust[i] ,d_LingerCount[i] , i);
    CUDA_CHECK_ERROR(); 
  }

}





// bug here adress in cuda 
void Escape_ReceiveBuffer_From_SendBuffer(AtomInfo **ReceBuffer , AtomInfo **SendBuffer , int **h_Escape_SendCount , int deviceCount ,int SegInterval )
{
    for(int i=0;i<deviceCount;i++) 
  { 
      for(int j=0;j<deviceCount;j++)
     {
         if(j != i)
        {
          
          CUDA_CALL( hipMemcpyPeerAsync( &(ReceBuffer[j][i*SegInterval]) , j , &(SendBuffer[i][j*SegInterval]) , i , sizeof(AtomInfo) * h_Escape_SendCount[i][j] )  );
          //CUDA_CALL( hipMemcpyPeer( &(ReceBuffer[j][i*SegInterval]) , j , &(SendBuffer[i][j*SegInterval]) , i , sizeof(AtomInfo) * h_Escape_SendCount[i][j] )  );
        } // 串行通信模拟
     }
  } 

}


// int ** h_Escape_ReceCount 
void Escape_Insert_ReceBuffer_Fellow_AtomInfoArray(AtomInfo **d_atoms , AtomInfo **ReceBuffer , int ** h_Escape_ReceCount , int **h_LingerCount , int* h_ParticleCount , int SegInterval ,int deviceCount )
{
    for(int i=0;i<deviceCount;i++)
    {
      int offset_atomArry = 0;
       for(int j=0;j<deviceCount;j++)
      {
        if(j != i)
        {
          hipSetDevice(i);
          CUDA_CALL( hipMemcpyAsync( &(d_atoms[i][h_LingerCount[i][0] + offset_atomArry]) , &(ReceBuffer[i][SegInterval * j]) , sizeof(AtomInfo) * h_Escape_ReceCount[i][j] , hipMemcpyDeviceToDevice)  );
          //CUDA_CALL( hipMemcpy( &(d_atoms[i][h_LingerCount[i][0] + offset_atomArry]) , &(ReceBuffer[i][SegInterval * j]) , sizeof(AtomInfo) * h_Escape_ReceCount[i][j] , hipMemcpyDeviceToDevice)  );
          offset_atomArry = offset_atomArry + h_Escape_ReceCount[i][j];
        }

      }

      h_ParticleCount[i] = h_LingerCount[i][0] + offset_atomArry;
      
    } //串行通信模式

} 







__global__ void test_Escape_Insert_kernel(AtomInfo *d_atoms, AtomInfo *ReceBuffer ,int * d_ReceCount ,int dev_id , int deviceCount,int SegInterval, int addShellCount )
{
  int id = threadIdx.x + blockIdx.x * blockDim.x;  // this atom's id
  if(id==0) // only one thread read data
  {
    if(dev_id != (deviceCount-1))
    {

      if( d_atoms[addShellCount-1].position[0] == ReceBuffer[ SegInterval*(deviceCount-1) + d_ReceCount[deviceCount-1] - 1].position[0])
         printf("the %dth Insert sucessfully !\n",dev_id); 
      else
         printf("the %dth Insert error !\n",dev_id);
    }
    else
    {

      if(d_atoms[addShellCount - 1].position[0] == ReceBuffer[ SegInterval*(deviceCount-1-1) + d_ReceCount[deviceCount-1-1]-1].position[0])
         printf("the %dth Insert sucessfully !\n",dev_id); 
      else
         printf("the %dth Insert error !\n",dev_id);

    }

    
  }  


}



void Check_Escape_Insert_ReceBuffer_Fellow_AtomInfoArray(AtomInfo **d_atoms , AtomInfo **ReceBuffer , int ** d_BuildShell_ReceCount  ,int *h_AddShell_ParticleCount , int SegInterval ,int deviceCount )
{
  for(int i=0;i<deviceCount;i++)
  {
    dim3 testBlock = 128;
    dim3 testGrid = 1;
    hipSetDevice(i);
    test_Escape_Insert_kernel<<<testGrid,testBlock>>>(d_atoms[i], ReceBuffer[i] , d_BuildShell_ReceCount[i] , i ,  deviceCount, SegInterval, h_AddShell_ParticleCount[i] );
    CUDA_CHECK_ERROR(); 
  }

}




