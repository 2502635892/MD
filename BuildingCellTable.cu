#include "hip/hip_runtime.h"
void GlobalCellTable_Set_Zero(int **GlobalCellTable,TableInfo T_Info,int deviceCount)
{
    for(int i=0;i<deviceCount;i++)
  {
     hipSetDevice(i);
     CUDA_CALL( hipMemset( GlobalCellTable[i] , 0 , sizeof(int) * Max_Mem * T_Info.Total_cellNUM ) );  // initialize element = 0
  }  // hipMemset 同步调用
 
}




// mapping the atom to local cell table kernel
__global__ void MapCellTable_Kernel(AtomInfo *d_atoms, int N, int *d_C_T, TableInfo T_Info)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    //int x,y,z;  // atom position

    double x,y,z;

    int Cell_coordinate_x;
    int Cell_coordinate_y;
    int Cell_coordinate_z;

    int cell_id;
    int count;

     if( id < N )
    {
       x = d_atoms[id].position[0];
       y = d_atoms[id].position[1];
       z = d_atoms[id].position[2]; // get the atom position

       Cell_coordinate_x = (int)(x / T_Info.L_cell[0]);
       Cell_coordinate_y = (int)(y / T_Info.L_cell[1]);
       Cell_coordinate_z = (int)(z / T_Info.L_cell[2]);   // get the  idth atom  coordinate

       cell_id = Cell_coordinate_x + Cell_coordinate_y * T_Info.cell_num[0] + Cell_coordinate_z * T_Info.cell_num[0] * T_Info.cell_num[1];

       count = atomicAdd( &d_C_T[Max_Mem * cell_id  + Max_Mem - 1] , 1);  // get the index of the  cell_id's th cell!!!

       d_C_T[Max_Mem * cell_id  + count ] = id;   // mapping the id th atom to the cell_id's th cell!
  
    }


}



void Mapping_to_CellTable(AtomInfo **d_atoms , int **GlobalCellTable , TableInfo T_Info , int *h_AddShell_ParticleCount , int deviceCount)
{
     for(int i=0;i<deviceCount;i++)
    {
      int dimgrid;
      dim3 dimBlock = 128;
      int N =  h_AddShell_ParticleCount[i];  // subdomain has atom number(including shell cell atom)
      if (N % 128 == 0)
        dimgrid = N / 128;
      else
        dimgrid = N / 128 + 1;

      dim3 dimGrid = dimgrid;

      hipSetDevice(i);
      MapCellTable_Kernel<<<dimGrid,dimBlock>>>( d_atoms[i], N, GlobalCellTable[i] , T_Info );
      CUDA_CHECK_ERROR();
    }   
  
    All_devices_Synchronize(deviceCount); 
}


