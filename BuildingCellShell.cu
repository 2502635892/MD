#include "hip/hip_runtime.h"

void BocellNumSubdomain(int *SubdomainBoSize,subDomain *subdomain,int Pnum)
{
	 int outtemp = 0;
	 int intemp = 0;
	 for(int i=0;i<Pnum;i++)
	{
		outtemp = (subdomain[i].X.high - subdomain[i].X.low + 3)* (subdomain[i].Y.high - subdomain[i].Y.low + 3)*(subdomain[i].Z.high - subdomain[i].Z.low + 3);
		intemp = (subdomain[i].X.high - subdomain[i].X.low + 1)* (subdomain[i].Y.high - subdomain[i].Y.low + 1)*(subdomain[i].Z.high - subdomain[i].Z.low + 1);
		SubdomainBoSize[i] = outtemp - intemp;
	}
	
}





void d_BuildShell_SendCount_Set_Zero(int **d_BuildShell_SendCount,int deviceCount)
{
    for(int i=0;i<deviceCount;i++)
    {
      hipSetDevice(i);
      CUDA_CALL( hipMemset(d_BuildShell_SendCount[i], 0 , sizeof(int)*deviceCount ) ); //set SendCount[i] to 0
    } 
   // hipMemset() 是同步调用函数
}


//  get the cell which subdomain it belong to and the sub domain id    ON GPU RUN
__device__ inline int d_cellToSubDomainId(int cx,int cy,int cz,TableInfo T_Info,deProcess P)
{
	int subDomain_id = -1;

    int Xcellnum = T_Info.cell_num[0];
	int Ycellnum = T_Info.cell_num[1];
	int Zcellnum = T_Info.cell_num[2];

	int XsubId = BLOCK_OWNER(cx,P.x,Xcellnum);
	int YsubId = BLOCK_OWNER(cy,P.y,Ycellnum);
	int ZsubId = BLOCK_OWNER(cz,P.z,Zcellnum);

    subDomain_id = XsubId + YsubId * P.x + ZsubId * P.x * P.y;
    return subDomain_id;
}




__global__ void Load_ShellAtoms_ToSendBuffer(AtomInfo *SubDomainAtomInfo,AtomInfo *SendBuffer,deProcess P,TableInfo T_Info,int subdomaimId,int Nsub,int Pnum,int *SendCount,int SegInterval)
{
   int id = threadIdx.x + blockIdx.x * blockDim.x;
   int cx,cy,cz;
   int neibArr[PNUM];
   for(int i=0;i<Pnum;i++)
      neibArr[i] = 0;
      
   int xyzId = -1;
   int pbc_x;
   int pbc_y;
   int pbc_z;
   int i_offset ;
   
   if(id < Nsub)
   {
     //  get the atom cell coordinate
     cx = (int)(SubDomainAtomInfo[id].position[0] / T_Info.L_cell[0]);
     cy = (int)(SubDomainAtomInfo[id].position[1] / T_Info.L_cell[1]);
     cz = (int)(SubDomainAtomInfo[id].position[2] / T_Info.L_cell[2]);
    // Boundary Cell to Neibor Domain
	  for(int i=cx-1;i<=cx+1;i++)
	{
		  for(int j=cy-1;j<=cy+1;j++)
		{
		     for(int k=cz-1;k<=cz+1;k++)
		   {
              // pbc boundary 
		   	      pbc_x = (i + T_Info.cell_num[0]) % T_Info.cell_num[0];
              pbc_y = (j + T_Info.cell_num[1]) % T_Info.cell_num[1];
              pbc_z = (k + T_Info.cell_num[2]) % T_Info.cell_num[2]; 
              
			       xyzId = d_cellToSubDomainId(pbc_x ,pbc_y ,pbc_z ,T_Info ,P );
		   	  
		   	   if(xyzId != subdomaimId)
		   	  {
		   	  	neibArr[xyzId] = 1;   //tag which subDomain should send to
			  }
		   	    
		   }	
		}
	}
    
       for(int i=0;i<Pnum;i++)
      {
	       if(neibArr[i] != 0)
	     {
	        i_offset = atomicAdd(&SendCount[i],1); // get the index atomicly 

	        if( i_offset < SegInterval )
	        {
             SendBuffer[ SegInterval * i + i_offset ] = SubDomainAtomInfo[id];
	        }
	        else
	        {
	          printf("out of buffer\n");
	        }
	          
	     }
	 }
       
  } // endif 
  
}//no bug surely





void Load_Shell_Atoms_To_SendBuffer(AtomInfo ** d_atoms,int* h_ParticleCount,AtomInfo **SendBuffer,int deviceCount,deProcess P,TableInfo T_Info ,int Pnum,int **d_BuildShell_SendCount,int SegInterval)
{
     for(int i=0;i<deviceCount;i++) 
    { 
      int N = h_ParticleCount[i];   
      int dimgrid;
      dim3 dimBlock = 128;
      if (N % 128 == 0)
        dimgrid = N / 128;
      else
        dimgrid = N / 128 + 1;
	    dim3 dimGrid = dimgrid;
	  
      hipSetDevice(i);
	    Load_ShellAtoms_ToSendBuffer<<<dimGrid,dimBlock>>>(d_atoms[i] ,SendBuffer[i] ,P ,T_Info ,i ,N ,Pnum ,d_BuildShell_SendCount[i] ,SegInterval );
      CUDA_CHECK_ERROR();
    } 

    All_devices_Synchronize(deviceCount);    // load data Synchronize！
  
}



void d_BuildShell_SendCount_DeviceToHost(int **d_BuildShell_SendCount,int ** h_BuildShell_SendCount, int deviceCount)
{
     for(int i=0;i<deviceCount;i++)
    {
      hipSetDevice(i);
      CUDA_CALL( hipMemcpy(h_BuildShell_SendCount[i] , d_BuildShell_SendCount[i] ,sizeof(int) * deviceCount,hipMemcpyDeviceToHost) );
    }
}






__global__ void test_loadShell_kernel(AtomInfo *d_atoms, int *SendCount,int deviceCount,int SegInterval, int thisdevice) // N : total !  count: check element number!
{
  int id = threadIdx.x + blockIdx.x * blockDim.x;  // this atom's id
  if(id==0) // only one thread read data
  {
     printf("sendBuffer check the first 、the last and last next element:\n");
    
     int i;
     for(int j=0;j<deviceCount;j++)
     {
      printf("%d \n",SendCount[j]);
      if(j != thisdevice)
      {
        i = SegInterval * j +  SendCount[j];
        printf("%d:%f %f %f \t %d:%f %f %f \t %d:%f %f %f \n",SegInterval * j,
        d_atoms[0].position[0],d_atoms[0].position[1],d_atoms[0].position[2],
        i-1,d_atoms[i-1].position[0],d_atoms[i-1].position[1],d_atoms[i-1].position[2],
        i,d_atoms[i].position[0],d_atoms[i].position[1],d_atoms[i].position[2]);
        printf("\n\n");

      }

     }
    
  }  

}




void check_Load_Shell_Atoms_To_SendBuffer(AtomInfo **SendBuffer,int ** d_BuildShell_SendCount,int deviceCount,int SegInterval)
{

  for(int i=0;i<deviceCount;i++) 
  { 
    dim3 testBlock = 128;
    dim3 testGrid = 1;
    hipSetDevice(i);
    test_loadShell_kernel<<<testGrid,testBlock>>>(SendBuffer[i], d_BuildShell_SendCount[i], deviceCount,SegInterval,i) ;
    CUDA_CHECK_ERROR();
  } 

}




// bug here adress in cuda 
void Build_Shell_ReceiveBuffer_From_SendBuffer(AtomInfo **ReceBuffer , AtomInfo **SendBuffer , int **h_BuildShell_SendCount , int deviceCount ,int SegInterval )
{
    for(int i=0;i<deviceCount;i++) 
  { 
      for(int j=0;j<deviceCount;j++)
     {
         if(j != i)
        {
          // CUDA_CALL( hipMemcpyPeer( &(ReceBuffer[j][i*SegInterval]) , j , &(SendBuffer[i][j*SegInterval]) , i , sizeof(AtomInfo) * h_BuildShell_SendCount[i][j] )  );
           //CUDA_CALL( hipMemcpyPeer( ReceBuffer[j] + i * SegInterval, j ,SendBuffer[i] + j * SegInterval , i , sizeof(AtomInfo) * h_BuildShell_SendCount[i][j] )  );
          CUDA_CALL( hipMemcpyPeerAsync( &(ReceBuffer[j][i*SegInterval]) , j , &(SendBuffer[i][j*SegInterval]) , i , sizeof(AtomInfo) * h_BuildShell_SendCount[i][j] )  );
        } // 串行通信模拟
     }
  } 

}



void Build_Shell_ReceiveCount_From_SendCount(int ** d_BuildShell_ReceCount,int **d_BuildShell_SendCount , int deviceCount)
{
  for(int i=0;i<deviceCount;i++)  // ith gpu send
  { 
      for(int j=0;j<deviceCount;j++)
     {
        CUDA_CALL( hipMemcpyPeer( &(d_BuildShell_ReceCount[i][j]) , i , &(d_BuildShell_SendCount[j][i]) , j , sizeof(int) )  );
     } // 串行通信
  } 

}



void Build_Shell_ReceCount_DeviceToHost(int ** h_BuildShell_ReceCount , int ** d_BuildShell_ReceCount , int deviceCount)
{
    for(int i=0;i<deviceCount;i++)
    {
      hipSetDevice(i);
      CUDA_CALL( hipMemcpy( h_BuildShell_ReceCount[i] , d_BuildShell_ReceCount[i] , sizeof(int) * deviceCount , hipMemcpyDeviceToHost ) );
    }
}




void Build_Shell_Insert_ReceBuffer_Fellow_AtomInfoArray(AtomInfo **d_atoms , AtomInfo **ReceBuffer , int ** h_BuildShell_ReceCount , int* h_ParticleCount ,int *h_AddShell_ParticleCount , int SegInterval ,int deviceCount )
{
    for(int i=0;i<deviceCount;i++)
    {
      int offset_atomArry = 0;
       for(int j=0;j<deviceCount;j++)
      {
        if(j != i)
        {
          hipSetDevice(i);
          CUDA_CALL( hipMemcpyAsync( &(d_atoms[i][h_ParticleCount[i] + offset_atomArry]) , &(ReceBuffer[i][SegInterval * j]) , sizeof(AtomInfo) * h_BuildShell_ReceCount[i][j] , hipMemcpyDeviceToDevice)  );
          //CUDA_CALL( hipMemcpy( &(d_atoms[i][h_ParticleCount[i] + offset_atomArry]) , &(ReceBuffer[i][SegInterval * j]) , sizeof(AtomInfo) * h_BuildShell_ReceCount[i][j] , hipMemcpyDeviceToDevice)  );
          offset_atomArry = offset_atomArry + h_BuildShell_ReceCount[i][j];
        }

      }

      h_AddShell_ParticleCount[i] = h_ParticleCount[i] + offset_atomArry;
      
    } //串行通信模式

} 





__global__ void test_Insert_kernel(AtomInfo *d_atoms, AtomInfo *ReceBuffer ,int * d_ReceCount ,int dev_id , int deviceCount,int SegInterval, int addShellCount )
{
  int id = threadIdx.x + blockIdx.x * blockDim.x;  // this atom's id
  if(id==0) // only one thread read data
  {
    if(dev_id != (deviceCount-1))
    {

      if( d_atoms[addShellCount-1].position[0] == ReceBuffer[ SegInterval*(deviceCount-1) + d_ReceCount[deviceCount-1] - 1].position[0])
         printf("the %dth Insert sucessfully !\n",dev_id); 
      else
         printf("the %dth Insert error !\n",dev_id);
    }
    else
    {

      if(d_atoms[addShellCount - 1].position[0] == ReceBuffer[ SegInterval*(deviceCount-1-1) + d_ReceCount[deviceCount-1-1]-1].position[0])
         printf("the %dth Insert sucessfully !\n",dev_id); 
      else
         printf("the %dth Insert error !\n",dev_id);

    }

    
  }  


}


void Check_Insert_ReceBuffer_Fellow_AtomInfoArray(AtomInfo **d_atoms , AtomInfo **ReceBuffer , int ** d_BuildShell_ReceCount  ,int *h_AddShell_ParticleCount , int SegInterval ,int deviceCount )
{
  for(int i=0;i<deviceCount;i++)
  {
    dim3 testBlock = 128;
    dim3 testGrid = 1;
    hipSetDevice(i);
    test_Insert_kernel<<<testGrid,testBlock>>>(d_atoms[i], ReceBuffer[i] , d_BuildShell_ReceCount[i] , i ,  deviceCount, SegInterval, h_AddShell_ParticleCount[i] );
    CUDA_CHECK_ERROR(); 
  }

}





