#include "hip/hip_runtime.h"
//  ==================force computation kernel================

__global__ void Getforce_Potential_kernel(AtomInfo *d_atoms ,int N ,int *d_C_T ,TableInfo T_Info ,double *d_U ,int *U_lock )
{
  __shared__ double local_d_U[128];

  double f[Dim] = {0.0, 0.0, 0.0};
  double r12[Dim] = {0.0, 0.0, 0.0};
  double d12_square = 0.0;
  double d_6 = 0.0;
  double d_8 = 0.0;
  double d_12 = 0.0;
  double d_14 = 0.0;
  double f12 = 0.0;
  //double sigma_6 = pow(SIGMA, 6);
  //double sigma_12 = sigma_6 * sigma_6;
  int Cell_coordinate[Dim] = {0, 0, 0};
  int cell_id = 0;

  //int i = 0;
  int j = 0;
  int ci = 0;
  int cj = 0;
  int ck = 0;

  int neibor_x = 0;
  int neibor_y = 0;
  int neibor_z = 0;

  double Imag_x = 0.0;
  double Imag_y = 0.0;
  double Imag_z = 0.0;

  double Box_Lx = L_x * a_x;
  double Box_Ly = L_y * a_y;
  double Box_Lz = L_z * a_z; // the simulation area Box size

  double k_x = 0.0;
  double k_y = 0.0;
  double k_z = 0.0;

  int id = threadIdx.x + blockIdx.x * blockDim.x;  // this atom's id
  local_d_U[threadIdx.x] = 0.0;                   // location potential energy

  if (id < N)
  {

    Cell_coordinate[0] = (int)(d_atoms[id].position[0] / T_Info.L_cell[0]); // locatied the idth atom in which cell
    Cell_coordinate[1] = (int)(d_atoms[id].position[1] / T_Info.L_cell[1]);
    Cell_coordinate[2] = (int)(d_atoms[id].position[2] / T_Info.L_cell[2]); //buge two how to map int the right cell

    f[0] = 0.0;
    f[1] = 0.0;
    f[2] = 0.0;

    for (ci = Cell_coordinate[0] - 1; ci <= Cell_coordinate[0] + 1; ci++)
    {
      for (cj = Cell_coordinate[1] - 1; cj <= Cell_coordinate[1] + 1; cj++)
      {
        for (ck = Cell_coordinate[2] - 1; ck <= Cell_coordinate[2] + 1; ck++)
        {
          neibor_x = (ci + T_Info.cell_num[0]) % T_Info.cell_num[0];
          neibor_y = (cj + T_Info.cell_num[1]) % T_Info.cell_num[1];
          neibor_z = (ck + T_Info.cell_num[2]) % T_Info.cell_num[2]; 
                                                        //perodical bound condition(PBC)
          cell_id = neibor_x + neibor_y * T_Info.cell_num[0] + neibor_z * T_Info.cell_num[0] * T_Info.cell_num[1]; //get neibor cell id

          for (j = 0; j < d_C_T[cell_id * Max_Mem + Max_Mem - 1]; j++)
          {
            if (d_C_T[cell_id * Max_Mem + j] != id)
            {

              k_x = floor((double)(ci) / (double)T_Info.cell_num[0]);

              k_y = floor((double)(cj) / (double)T_Info.cell_num[1]);

              k_z = floor((double)(ck) / (double)T_Info.cell_num[2]);

              Imag_x = k_x * Box_Lx + d_atoms[d_C_T[cell_id * Max_Mem + j]].position[0];
              Imag_y = k_y * Box_Ly + d_atoms[d_C_T[cell_id * Max_Mem + j]].position[1];
              Imag_z = k_z * Box_Lz + d_atoms[d_C_T[cell_id * Max_Mem + j]].position[2]; // pbc

              r12[0] = Imag_x - d_atoms[id].position[0];
              r12[1] = Imag_y - d_atoms[id].position[1];
              r12[2] = Imag_z - d_atoms[id].position[2];

              d12_square = r12[0] * r12[0] + r12[1] * r12[1] + r12[2] * r12[2];

              d_6 = d12_square * d12_square * d12_square;
              d_8 = d12_square * d_6;
              d_12 = d_6 * d_6;
              d_14 = d_6 * d_8;

              f12 = (sigma_6 / d_8 - 2.0 * sigma_12 / d_14) * 24.0 * EPSILON; // LJ coeffiencient for force

              f[0] = f[0] + f12 * r12[0];
              f[1] = f[1] + f12 * r12[1];
              f[2] = f[2] + f12 * r12[2];

              local_d_U[threadIdx.x] = local_d_U[threadIdx.x] + 4.0 * EPSILON * (sigma_12 / d_12 - sigma_6 / d_6);
            }
          }
        }
      }
    }

    d_atoms[id].force[0] = f[0];
    d_atoms[id].force[1] = f[1];
    d_atoms[id].force[2] = f[2];

  } // the Force of atom culculation

  __syncthreads();

  if ((id < N) && (threadIdx.x == 0))
  {
    int i;
    double block_U_sum = 0.0;
    for (i = 0; i < blockDim.x; i++)
    {
      block_U_sum = block_U_sum + local_d_U[i];
    }

    while (atomicCAS(U_lock, 0, 1) != 0)
      ; // lock()

    *d_U = *d_U + block_U_sum;

     atomicExch(U_lock, 0); // unlock()
  }

} // get the d_atoms[id] force and energy !!!



// force computation !!!   
void Computation_Force_Potential(AtomInfo **d_atoms ,int* h_ParticleCount ,int **GlobalCellTable , TableInfo T_Info ,double **d_U , int **U_lock , int deviceCount)
{
    for(int i=0;i<deviceCount;i++)
    {
      int dimgrid;
      dim3 dimBlock = 128;
      int N =  h_ParticleCount[i];  // subdomain has atom number
      if (N % 128 == 0)
        dimgrid = N / 128;
      else
        dimgrid = N / 128 + 1;
    
      dim3 dimGrid = dimgrid;
      hipSetDevice(i);
      Getforce_Potential_kernel<<<dimGrid,dimBlock>>>(d_atoms[i] , N , GlobalCellTable[i], T_Info, d_U[i], U_lock[i] );
      CUDA_CHECK_ERROR();
    }
    //设备执行同步
    All_devices_Synchronize(deviceCount); 
}



void d_U_Set_Zero(double **d_U , int deviceCount)
{
   for(int i=0;i<deviceCount;i++)
  {
     hipSetDevice(i);
     CUDA_CALL( hipMemset(d_U[i], 0, sizeof(double) ) ); //set U_lock[i] to 0
  }
  
}


double Get_Total_Potential(double **h_U, double **d_U, int deviceCount)
{
     for(int i=0;i<deviceCount;i++)
    {
      hipSetDevice(i);
      CUDA_CALL( hipMemcpy(h_U[i],d_U[i] ,sizeof(double) ,hipMemcpyDeviceToHost) );
    }

    double h_total_U = 0.0;
    
     for(int i=0;i<deviceCount;i++)
    {
      h_total_U += h_U[i][0];
    }

    return h_total_U;
}