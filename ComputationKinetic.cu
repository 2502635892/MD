#include "hip/hip_runtime.h"
__global__ void GetKinetic_Kernel(AtomInfo* d_atoms ,int N ,double* d_EK ,int *Signal )
{
    __shared__ double per_kinetic[128];
    int id = threadIdx.x +  blockIdx.x * blockDim.x; // this atom's id 
    per_kinetic[threadIdx.x] = 0.0;

     if (id < N)
    {
      per_kinetic[threadIdx.x] = 0.5 * d_atoms[id].mass * (d_atoms[id].velocity[0] * d_atoms[id].velocity[0] + d_atoms[id].velocity[1] * d_atoms[id].velocity[1] + d_atoms[id].velocity[2] * d_atoms[id].velocity[2]);
    }
    __syncthreads();

    if ((id < N) && (threadIdx.x == 0)) // block add to Ek
  {
     int i;
     double Block_kinetic_sum = 0.0;
     for (i = 0; i < blockDim.x; i++)
    {
      Block_kinetic_sum = Block_kinetic_sum + per_kinetic[i];
    }

     while( atomicCAS(Signal,0,1) != 0 ) ;   // lock()
       *d_EK = *d_EK + Block_kinetic_sum ;  // add the Block_kinetic_sum to d_EK
     atomicExch(Signal,0);  // unlock() 
  }

}


// computation the kenetic 
void Computation_Kinetic(AtomInfo **d_atoms ,int* h_ParticleCount, double **d_EK, int **Signal, int deviceCount)
{
   for(int i=0;i<deviceCount;i++)
 {
    int dimgrid;
    dim3 dimBlock = 128;
    int N = h_ParticleCount[i];  // subdomain has atom number

    if (N % 128 == 0)
      dimgrid = N / 128;
    else
      dimgrid = N / 128 + 1;

    dim3 dimGrid = dimgrid;

    hipSetDevice(i);
    GetKinetic_Kernel<<<dimGrid,dimBlock>>>(d_atoms[i], N, d_EK[i], Signal[i]);
    CUDA_CHECK_ERROR();

 }
  // 计算动能 同步！
  All_devices_Synchronize(deviceCount); 

}



void d_Ek_Set_Zero(double **d_EK, int deviceCount)
{
    for(int i=0;i<deviceCount;i++)
  {
     hipSetDevice(i);
     CUDA_CALL( hipMemset(d_EK[i], 0, sizeof(double) ) ); //set d_EK[i] to 0
  }
  
}



double Get_Total_Kinetic(double **h_EK, double **d_EK, int deviceCount)
{
   for(int i=0;i<deviceCount;i++)
  {
    hipSetDevice(i);
    CUDA_CALL( hipMemcpy(h_EK[i],d_EK[i] ,sizeof(double) ,hipMemcpyDeviceToHost) );
  }

  double total_EK = 0.0;
   for(int i=0;i<deviceCount;i++)
  {
     total_EK += h_EK[i][0];
     //printf("\n%dth GPU:%f\t\n",i,h_EK[i][0]);
  }

  return total_EK;
}