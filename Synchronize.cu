#include "hip/hip_runtime.h"
void  All_devices_Synchronize(int deviceCount)
{
    for(int i=0;i<deviceCount;i++)
   {
     hipSetDevice(i);
     CUDA_CALL(hipDeviceSynchronize());
   } 
}



__global__ void test_atomticADD_kernel(int *d_S,int N)
{
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if(id < N)
  {
    int temp = atomicAdd(d_S,1);
    printf("%dthead: %d\n",id,temp);
  }

}

void test_atomticADD()
{
  dim3 testBlock = 64;
  dim3 testGrid = 2;
  int N =  64 * 2;
  hipSetDevice(1);

  int *d_S;
  int *h_S = (int*)malloc(sizeof(int));
  CUDA_CALL( hipMalloc( &d_S, sizeof(int) ) );
  CUDA_CALL( hipMemset(d_S , 0 , sizeof(int) ) );

  test_atomticADD_kernel<<<testGrid,testBlock>>>(d_S, N);
  CUDA_CHECK_ERROR();

  CUDA_CALL( hipMemcpy(h_S,d_S ,sizeof(int) ,hipMemcpyDeviceToHost) );
  printf("%d\n",*h_S);

}


__global__ void test_cudaMalloc_kernel(int *d_p ,int N)
{
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if(id==0)
  {
    for(int i=0;i<N;i++)
      printf("%d %d %d %d %d %d\n", d_p[i] , *(d_p + i) , &d_p[i] , (d_p + i) , &(*(d_p + i)) , d_p );
  }

}



void test_cudaMalloc()
{
  int *d_p;
  int N = 10;

  hipSetDevice(0);
  int *p = (int*)malloc(sizeof(int)*N);
  for(int i=0;i<N;i++)
     p[i] = i; 
  
  CUDA_CALL( hipMalloc(&d_p,sizeof(int)*N) );
  CUDA_CALL( hipMemcpy(d_p , p ,sizeof(int) * N ,hipMemcpyHostToDevice) );

  dim3 testBlock = 64;
  dim3 testGrid = 1;
  test_cudaMalloc_kernel<<<testGrid,testBlock>>>(d_p,N);


}
