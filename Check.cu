
void check_copy( AtomInfo *check_atoms , AtomInfo **d_atoms , int* h_ParticleCount , int deviceCount )
{
    int offset = 0;
    for(int i=0;i<deviceCount;i++)
    {
      hipSetDevice(i);
      CUDA_CALL(hipMemcpy( &check_atoms[offset] , d_atoms[i] , sizeof(AtomInfo) * h_ParticleCount[i] , hipMemcpyDeviceToHost ) );
      offset = offset + h_ParticleCount[i];
    } 

}


void Sort_atoms(AtomInfo *check_atoms , int N,int step, int deviceCount)
{
    AtomInfo *Sort_atoms = (AtomInfo *)malloc(sizeof(AtomInfo) * N); 
    
    for(int i=0;i<N;i++)
    {
        Sort_atoms[check_atoms[i].atom_id] = check_atoms[i] ; 
    }
    
    char file[1024];
    sprintf(file,"%d_%d_step_check.data",deviceCount,step);
    FILE *fp = fopen(file, "wb");


    for(int i=0;i<N;i++)
    {
        // V23// fprintf(fp,"%d\t%lf\t%lf\t%lf\t\n", i , Sort_atoms[i].position[0], Sort_atoms[i].position[1] , Sort_atoms[i].position[2]);
        // V24// fprintf(fp,"%d\t%lf\t%lf\t%lf\t\n", i , Sort_atoms[i].velocity[0], Sort_atoms[i].velocity[1] , Sort_atoms[i].velocity[2]);
       // fprintf(fp,"%d\t%lf\t%lf\t%lf\t\n", i , Sort_atoms[i].force[0], Sort_atoms[i].force[1] , Sort_atoms[i].force[2]);
        fprintf(fp,"%d\t%lf\t%lf\t%lf\t%lf\t%lf\t%lf\t\n", i , Sort_atoms[i].position[0], Sort_atoms[i].position[1] , Sort_atoms[i].position[2],
        Sort_atoms[i].velocity[0], Sort_atoms[i].velocity[1] , Sort_atoms[i].velocity[2]);
        
    }
      


    free(Sort_atoms);
    fclose(fp);

}





