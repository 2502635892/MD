#include "hip/hip_runtime.h"
void MapParticleToSubdomain(AtomInfo *atoms,AtomInfo **atomsInfoArr,int MallocLen,TableInfo T_Info,deProcess P,int* h_ParticleCount,int n,int Pnum)
{
	double x,y,z;
	int cx,cy,cz;
	int subDomainId = -1;
	
	  for(int i=0; i<n;i++)  // n is the total atom number in the system-
	{
	    x = atoms[i].position[0];
	  	y = atoms[i].position[1];
	  	z = atoms[i].position[2];
	  	
		cx = (int)(x / T_Info.L_cell[0]);
		cy = (int)(y / T_Info.L_cell[1]);
		cz = (int)(z / T_Info.L_cell[2]); 
		
		subDomainId = GetCellSubDomainId(cx,cy,cz,T_Info,P);
		
        atomsInfoArr[subDomainId][h_ParticleCount[subDomainId]] = atoms[i]; // insert the atom to the right subdomain 
    
        h_ParticleCount[subDomainId]++;   // the counter add 1
    
		if( h_ParticleCount[subDomainId] >= MallocLen ) // out of memory check !!!
		{
			printf("the subdomain atom out of the buffer!\n");
			exit(1);
		}
		
	}
	

}



void Check_MapParticleToSubdomain(AtomInfo **atomsInfoArr,TableInfo T_Info,deProcess P,int* h_ParticleCount,int Pnum)
{
   int Particle_sum = 0;

   for(int i=0;i<Pnum;i++ )
  {
      Particle_sum = Particle_sum + h_ParticleCount[i];
      int count = 0;
      for(int j=0;j<h_ParticleCount[i];j++)
      {
         double x = atomsInfoArr[i][j].position[0];
         double y = atomsInfoArr[i][j].position[1];
         double z = atomsInfoArr[i][j].position[2];

         int cx = (int)(x / T_Info.L_cell[0]);
		 int cy = (int)(y / T_Info.L_cell[1]);
		 int cz = (int)(z / T_Info.L_cell[2]); 
         
         int subDomainId = GetCellSubDomainId(cx,cy,cz,T_Info,P);
         if(subDomainId == i)
         {
            count++;
         }  else{
              printf("the %dth partition error\n ",i);
         }


      }

      if(count ==h_ParticleCount[i])
        printf("%dth Partition OK!\n",i);
      printf("check the first the last and the last next element:\n");
      printf("%f %f %f \t %f %f %f \t %f %f %f \n",atomsInfoArr[i][0].position[0], atomsInfoArr[i][0].position[1], atomsInfoArr[i][0].position[2]
      ,atomsInfoArr[i][h_ParticleCount[i]-1].position[0], atomsInfoArr[i][h_ParticleCount[i]-1].position[1], atomsInfoArr[i][h_ParticleCount[i]-1].position[2] ,
      atomsInfoArr[i][h_ParticleCount[i]].position[0], atomsInfoArr[i][h_ParticleCount[i]].position[1], atomsInfoArr[i][h_ParticleCount[i]].position[2]);  
          
  }

  printf("the total Particel is %d\n",Particle_sum);

}





// -------------memeroy testing kernel----------------
__global__ void test_kernel(AtomInfo *d_atoms, int N) // N : total !  count: check element number!
{
  
  int id = threadIdx.x + blockIdx.x * blockDim.x;  // this atom's id
  if(id==0) // only one thread read data
  {
     printf("check the first 、the last and last next element:\n");
     int  i = N;
     printf("%f %f %f \t %f %f %f \t %f %f %f \n",
         d_atoms[0].position[0],d_atoms[0].position[1],d_atoms[0].position[2],
         d_atoms[i-1].position[0],d_atoms[i-1].position[1],d_atoms[i-1].position[2],
		     d_atoms[i].position[0],d_atoms[i].position[1],d_atoms[i].position[2]);

    printf("\n\n");

  }  

}


void test_Copy(AtomInfo **d_atoms,int *h_ParticleCount,int deviceCount)
{
  //=========== check the force computation =======
  for(int i=0;i<deviceCount;i++)
 { 
    dim3 testBlock = 128;
    dim3 testGrid = 1;
    int N =  h_ParticleCount[i];
  
    hipSetDevice(i);
    test_kernel<<<testGrid,testBlock>>>(d_atoms[i],N);
    CUDA_CHECK_ERROR();
 }

}

