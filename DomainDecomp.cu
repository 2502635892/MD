void GetTableInfo(TableInfo &T_Info)
{
  double Box_Lx = L_x * a_x;
  double Box_Ly = L_y * a_y;
  double Box_Lz = L_z * a_z; // the simulation area Box size
  
  T_Info.cell_num[0] = (int)(Box_Lx / RCUT);
  T_Info.cell_num[1] = (int)(Box_Ly / RCUT);
  T_Info.cell_num[2] = (int)(Box_Lz / RCUT); // the cell divide on x ,y ,z direction & the number of cell bins

  T_Info.L_cell[0] = Box_Lx / (double)T_Info.cell_num[0];
  T_Info.L_cell[1] = Box_Ly / (double)T_Info.cell_num[1];
  T_Info.L_cell[2] = Box_Lz / (double)T_Info.cell_num[2]; // get the cell length

  T_Info.Total_cellNUM = T_Info.cell_num[0] * T_Info.cell_num[1] * T_Info.cell_num[2];
}



//-------------------------make the segment partition------------
void segPartiton(seg *segPoint,int segCount,int n)
{
      for(int i=0 ;i < segCount; i++)
    {
      segPoint[i].low = BLOCK_LOW(i,segCount,n);
      segPoint[i].high = BLOCK_HIGH(i,segCount,n);
	 }

}


int GetCellSubDomainId(int cx,int cy,int cz,TableInfo T_Info,deProcess P)
{
    int subDomain_id = -1;

    int Xcellnum = T_Info.cell_num[0];
	int Ycellnum = T_Info.cell_num[1];
	int Zcellnum = T_Info.cell_num[2];

	int XsubId = BLOCK_OWNER(cx,P.x,Xcellnum);
	int YsubId = BLOCK_OWNER(cy,P.y,Ycellnum);
	int ZsubId = BLOCK_OWNER(cz,P.z,Zcellnum);

    subDomain_id = XsubId + YsubId * P.x + ZsubId * P.x * P.y;
    return subDomain_id;

}



void check_GetCellSubDomainId(TableInfo T_Info,subDomain *subdomain,deProcess P)
{
  int count = 0;
  for(int i=0;i<T_Info.cell_num[2];i++ )
  {
      for(int j = 0;j< T_Info.cell_num[1];j++)
      { 
          for(int k=0;k<T_Info.cell_num[0];k++)
          {
              int id =  GetCellSubDomainId( k , j , i ,T_Info, P);
              bool b1 = ( k>= subdomain[id].X.low) && (k<=subdomain[id].X.high);
              bool b2 = ( j>= subdomain[id].Y.low) && (j<=subdomain[id].Y.high);
              bool b3 = ( i>= subdomain[id].Z.low) && (i<=subdomain[id].Z.high);
              bool b = ( (b1&&b2) && b3);
              if(b)
              {
                 count++;
              }else
              {
                  printf("error!!\n");
              }

      }

  }

}

if(count == T_Info.cell_num[0]*T_Info.cell_num[1]*T_Info.cell_num[2])
  printf("check is ok\n");


}