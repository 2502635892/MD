#include "def.h"
#include "dataStruct.h"
#include "initial.c"
#include "Synchronize.cu"
#include "DomainDecomp.cu"
#include "BuildingCellShell.cu"
#include "ParticlePartition.cu"
#include "BuildingCellTable.cu"
#include "ComputationForcePotential.cu"
#include "VerletIntegration.cu"
#include "ComputationKinetic.cu"
#include "EscapeAdjust.cu"
#include "Check.cu"

void md_lj(void)
{
  int n0 = FCC_NUM;
  int nxyz[3] = {L_x, L_y, L_z};
  double a[3] = {a_x, a_y, a_z};
  double T = Termp;
  int N = n0 * nxyz[0] * nxyz[1] * nxyz[2];  // total atom number in the system 
  double dt = DT;   // integration step == DT==5
  deProcess P;      // process x,y,z
  TableInfo T_Info; // table info
  // process decompoment
  P.x = 2;
  P.y = 1;
  P.z = 1;

  int Pnum = P.x * P.y * P.z; //  Pnum : process numbers !

  AtomInfo *atoms = (AtomInfo *)malloc(sizeof(AtomInfo) * N);  // malloc the memory for the total atom 
  initialize(atoms, N, nxyz, a, T);   //initialize the atoms information
  GetTableInfo(T_Info);               // get the cell divided information

  // --------- segment malloc memeory-----------
  seg *Xseg = (seg *)malloc(sizeof(seg) * P.x);
  seg *Yseg = (seg *)malloc(sizeof(seg) * P.y);
  seg *Zseg = (seg *)malloc(sizeof(seg) * P.z);
  
  // ---------segment split three coordinates ------------
  segPartiton(Xseg ,P.x ,T_Info.cell_num[0]);
  segPartiton(Yseg ,P.y ,T_Info.cell_num[1]);
  segPartiton(Zseg ,P.z ,T_Info.cell_num[2]);

  printf("%d %d %d\n",T_Info.cell_num[0],T_Info.cell_num[1],T_Info.cell_num[2]);
  
  //==++++++++++++++++++++ subdomain seg info array++++++++++++++++++++++++
  subDomain *subdomain = (subDomain *)malloc(sizeof(subDomain) * Pnum);  //  subdomain seg info
  
  int index = 0;
    for(int i=0; i<P.z; i++)
  { 
      for(int j=0;j<P.y;j++)
    {
         for(int k=0;k<P.x;k++)
       {
           subdomain[index].X = Xseg[k];
           subdomain[index].Y = Yseg[j];
           subdomain[index].Z = Zseg[i];
           printf("%d subdom:X(%d,%d)\n",index,subdomain[index].X.low,subdomain[index].X.high);
           printf("%d subdom:Y(%d,%d)\n",index,subdomain[index].Y.low,subdomain[index].Y.high);
           printf("%d subdom:Z(%d,%d)\n",index,subdomain[index].Z.low,subdomain[index].Z.high);
		       index ++;
       }
       
	}
  
  } // subdomain partition 
  


// malloc the memory for the subdomain atomInfo array  on CPU
int ATOMINFO_Malloc_Len ;
ATOMINFO_Malloc_Len = N/Pnum * SCALER ;

AtomInfo **atomsInfoArr = (AtomInfo **)malloc(sizeof(AtomInfo*) * Pnum);   // atomsInfoArr
for(int i=0;i<Pnum;i++)
  atomsInfoArr[i] = (AtomInfo *)malloc( sizeof(AtomInfo)  *  ATOMINFO_Malloc_Len );

// malloc  for subdomain atom number counter
int* h_ParticleCount = (int*)malloc( sizeof(int) * Pnum );
memset(h_ParticleCount,0,sizeof(int)*Pnum); // atomInfo array continue number

int* h_AddShell_ParticleCount = (int*)malloc( sizeof(int) * Pnum );
memset(h_AddShell_ParticleCount,0,sizeof(int)*Pnum); // atomInfo array continue number


MapParticleToSubdomain(atoms,atomsInfoArr,ATOMINFO_Malloc_Len,T_Info, P, h_ParticleCount, N, Pnum);

for(int i=0;i<Pnum;i++)
   printf("%dth:%d\n",i,h_ParticleCount[i]);



// get the boundary cell number of the sub domain!
int * SubDomainBoundarySize = (int *)malloc(sizeof(int) * Pnum);
BocellNumSubdomain(SubDomainBoundarySize,subdomain,Pnum); 

// cell shell size decision the SendBuffer & receiveBuffer size 
int max_subdomainshell = 0;
for(int i=0;i<Pnum;i++)
{
 if (SubDomainBoundarySize[i] > max_subdomainshell )
  max_subdomainshell = SubDomainBoundarySize[i];
}

int STEP = max_subdomainshell * rh / 2; 

int RECEVIE_Malloc_Len ;
    RECEVIE_Malloc_Len = STEP * Pnum ;

int SEND_Malloc_Len ;
    SEND_Malloc_Len =  STEP * Pnum ;

 // -------------Get the GPUs count number--------------------
 int deviceCount;
 CUDA_CALL( hipGetDeviceCount(&deviceCount) );
 printf("total GPU number is  %d  in system\n",deviceCount);

 deviceCount = Pnum;

 
 // Reset device
 for(int i=0;i<deviceCount;i++)
  {
    hipSetDevice(i);
    hipDeviceReset();
  } 

  // malloc memory on GPU remember Free
  AtomInfo **d_atoms = (AtomInfo **)malloc(sizeof(AtomInfo*) * deviceCount) ;
   for(int i=0;i<deviceCount;i++)
  {
    hipSetDevice(i);
    CUDA_CALL( hipMalloc(&d_atoms[i], sizeof(AtomInfo) * ATOMINFO_Malloc_Len) ); 
  }

  // copy atomInfo to GPU memory from CPU 
   for(int i=0;i<deviceCount;i++)
  {
     hipSetDevice(i);
     CUDA_CALL( hipMemcpy(d_atoms[i], atomsInfoArr[i], sizeof(AtomInfo) * h_ParticleCount[i], hipMemcpyHostToDevice) );
  }



AtomInfo **SendBuffer = (AtomInfo **)malloc(sizeof(AtomInfo*) * deviceCount) ;
 for(int i=0;i<deviceCount;i++)
{
  hipSetDevice(i);                       // SEND_Malloc_Len =  STEP * Pnum
  CUDA_CALL( hipMalloc( &SendBuffer[i] , sizeof(AtomInfo) * SEND_Malloc_Len ) ); 
}

AtomInfo **ReceBuffer = (AtomInfo **)malloc(sizeof(AtomInfo*) * deviceCount);
  for(int i=0;i<deviceCount;i++)
{
   hipSetDevice(i);                     // RECEVIE_Malloc_Len = STEP * Pnum
   CUDA_CALL( hipMalloc(&ReceBuffer[i], sizeof(AtomInfo) * RECEVIE_Malloc_Len ) );  
}

int **d_BuildShell_SendCount = (int **)malloc( sizeof(int*) * deviceCount );
  for(int i=0;i<deviceCount;i++)
{
  hipSetDevice(i);
  CUDA_CALL( hipMalloc(&d_BuildShell_SendCount[i], sizeof(int)*deviceCount) );    
}

int ** h_BuildShell_SendCount = (int **)malloc(sizeof(int*) * deviceCount);
  for(int i=0;i<deviceCount;i++)
{
  h_BuildShell_SendCount[i] = (int *)malloc( sizeof(int)*deviceCount );
}

int ** d_BuildShell_ReceCount = (int **)malloc(sizeof(int*) * deviceCount);
  for(int i=0;i<deviceCount;i++)
{
  hipSetDevice(i);
  CUDA_CALL( hipMalloc(&d_BuildShell_ReceCount[i], sizeof(int)*deviceCount) );  
}

int ** h_BuildShell_ReceCount = (int **)malloc(sizeof(int*) * deviceCount);
  for(int i=0;i<deviceCount;i++)
{
  h_BuildShell_ReceCount[i] = (int *)malloc( sizeof(int)*deviceCount );
}

int **GlobalCellTable = (int **)malloc( sizeof(int*) * deviceCount);
  for(int i=0;i<deviceCount;i++)
{
   hipSetDevice(i);
   CUDA_CALL( hipMalloc( &GlobalCellTable[i], sizeof(int) * Max_Mem * T_Info.Total_cellNUM ) ); 
} 



// computation Force and potencial energy
double **d_U = (double **)malloc(sizeof(double*) * deviceCount);
  for(int i=0;i<deviceCount;i++)
 {
   hipSetDevice(i);
   CUDA_CALL( hipMalloc(&d_U[i], sizeof(double) ) );
 }

 int **U_lock = (int **)malloc(sizeof(int*) * deviceCount);
 for(int i=0;i<deviceCount;i++)
 {
   hipSetDevice(i);
   CUDA_CALL( hipMalloc(&U_lock[i], sizeof(int) ) );
   CUDA_CALL( hipMemset(U_lock[i], 0, sizeof(int) ) ); //set U_lock[i] to 0
 }

 double **h_U = (double **)malloc(sizeof(double*) * deviceCount);
 for(int i=0;i<deviceCount;i++)
 {
   h_U[i] = (double *)malloc( sizeof(double) );
 }




// malloc the d_EK for the kenetic computation
double **d_EK = (double **)malloc(sizeof(double*) * deviceCount);
for(int i=0;i<deviceCount;i++)
{
 hipSetDevice(i);
 CUDA_CALL( hipMalloc(&d_EK[i], sizeof(double) ) );
}

int **Signal = (int **)malloc( sizeof(int*) * deviceCount);
for(int i=0;i<deviceCount;i++)
{
 hipSetDevice(i);
 CUDA_CALL( hipMalloc( &Signal[i], sizeof(int) ) );
 CUDA_CALL( hipMemset(Signal[i], 0 , sizeof(int) ) ); //set d_EK[i] to 0
}

double **h_EK = (double **)malloc(sizeof(double*) * deviceCount);
 for(int i=0;i<deviceCount;i++)
{
 h_EK[i] = (double *)malloc( sizeof(double) );
}




// adjust array malloc on gpus
AtomInfo **d_Adjust = (AtomInfo **)malloc(sizeof(AtomInfo*) * deviceCount) ;
for(int i=0;i<deviceCount;i++)
{
  hipSetDevice(i);
  CUDA_CALL( hipMalloc(&d_Adjust[i], sizeof(AtomInfo) * ATOMINFO_Malloc_Len) ); // malloc memory on GPU remember Free
}


// Escape particle adjustment vars malloc
int **d_LingerCount = (int **)malloc(sizeof(int*) * deviceCount);
for(int i=0;i<deviceCount;i++)
{
  hipSetDevice(i);
  CUDA_CALL( hipMalloc( &d_LingerCount[i] , sizeof(int) ) ); // malloc memory on GPU remember Free
}

int **h_LingerCount = (int **)malloc( sizeof(int*) * deviceCount);
for(int i=0;i<deviceCount;i++)
{
  h_LingerCount[i] = (int*)malloc( sizeof(int) );
}


int **d_Escape_SendCount = (int **)malloc(sizeof(int*) * deviceCount);
  for(int i=0;i<deviceCount;i++)
{
  hipSetDevice(i);
  CUDA_CALL( hipMalloc(&d_Escape_SendCount[i], sizeof(int)*deviceCount) );    
}

int ** h_Escape_SendCount = (int **)malloc(sizeof(int*) * deviceCount);
  for(int i=0;i<deviceCount;i++)
{
  h_Escape_SendCount[i] = (int *)malloc( sizeof(int)*deviceCount );
}

int ** d_Escape_ReceCount = (int **)malloc(sizeof(int*) * deviceCount);
  for(int i=0;i<deviceCount;i++)
{
  hipSetDevice(i);
  CUDA_CALL( hipMalloc(&d_Escape_ReceCount[i], sizeof(int)*deviceCount) );  
}

int ** h_Escape_ReceCount = (int **)malloc(sizeof(int*) * deviceCount);
  for(int i=0;i<deviceCount;i++)
{
  h_Escape_ReceCount[i] = (int *)malloc( sizeof(int)*deviceCount );
}


double total_U = 0.0;
double total_Ek = 0.0;





// Mapping Cell Table on GPUS
d_BuildShell_SendCount_Set_Zero(d_BuildShell_SendCount , deviceCount );
Load_Shell_Atoms_To_SendBuffer(d_atoms , h_ParticleCount ,SendBuffer ,deviceCount ,P ,T_Info ,Pnum ,d_BuildShell_SendCount ,STEP );


d_BuildShell_SendCount_DeviceToHost(d_BuildShell_SendCount, h_BuildShell_SendCount, deviceCount);
Build_Shell_ReceiveBuffer_From_SendBuffer(ReceBuffer , SendBuffer , h_BuildShell_SendCount , deviceCount , STEP );


Build_Shell_ReceiveCount_From_SendCount( d_BuildShell_ReceCount, d_BuildShell_SendCount , deviceCount);
Build_Shell_ReceCount_DeviceToHost( h_BuildShell_ReceCount ,  d_BuildShell_ReceCount , deviceCount) ;
Build_Shell_Insert_ReceBuffer_Fellow_AtomInfoArray(d_atoms , ReceBuffer , h_BuildShell_ReceCount , h_ParticleCount , h_AddShell_ParticleCount , STEP , deviceCount );

GlobalCellTable_Set_Zero( GlobalCellTable , T_Info , deviceCount );
Mapping_to_CellTable( d_atoms , GlobalCellTable , T_Info , h_AddShell_ParticleCount , deviceCount);



// Computation Force and Potential on GPUs
 
 d_U_Set_Zero( d_U , deviceCount);
 Computation_Force_Potential(d_atoms , h_ParticleCount ,GlobalCellTable , T_Info , d_U , U_lock , deviceCount);
 
 
 total_U = Get_Total_Potential(h_U, d_U, deviceCount);

 // Get the Kinetic on GPUs
 d_Ek_Set_Zero(d_EK , deviceCount);
 Computation_Kinetic(d_atoms , h_ParticleCount, d_EK, Signal, deviceCount);
 total_Ek =  Get_Total_Kinetic(h_EK, d_EK, deviceCount);
 printf("kinetic is %f \t Potential is %f \n", total_Ek/N , 0.5 * total_U / N );



 int Ne = NE;
 int Np = NP;


   for(int step=1;step<=Ne+Np;step++)
 {
 
  
 // Update partition of Velocity and total Position
   Update_AtomsInfo_kernel_1(d_atoms , h_ParticleCount , dt , deviceCount);



// Escape particle Adjustment
d_LingerCount_Set_Zero( d_LingerCount, deviceCount);
d_Escape_SendCount_Set_Zero( d_Escape_SendCount , deviceCount);
Split_Escape_And_linger( d_atoms , h_ParticleCount , d_Adjust, d_LingerCount , SendBuffer, d_Escape_SendCount , deviceCount, P, T_Info , Pnum , STEP);


LingerCount_DeviceToHost(h_LingerCount , d_LingerCount, deviceCount);
linger_To_AtomInfoArray( d_atoms , d_Adjust , h_LingerCount , deviceCount);

d_Escape_SendCount_DeviceToHost(d_Escape_SendCount , h_Escape_SendCount , deviceCount);

d_Escape_ReceiveCount_From_SendCount(d_Escape_ReceCount,d_Escape_SendCount , deviceCount);
d_Escape_ReceCount_DeviceToHost(h_Escape_ReceCount , d_Escape_ReceCount , deviceCount);


Escape_ReceiveBuffer_From_SendBuffer(ReceBuffer,SendBuffer,h_Escape_SendCount, deviceCount , STEP );
Escape_Insert_ReceBuffer_Fellow_AtomInfoArray(d_atoms , ReceBuffer , h_Escape_ReceCount , h_LingerCount , h_ParticleCount ,STEP , deviceCount );


// Mapping Cell Table on GPUS
d_BuildShell_SendCount_Set_Zero(d_BuildShell_SendCount , deviceCount );
Load_Shell_Atoms_To_SendBuffer(d_atoms , h_ParticleCount ,SendBuffer ,deviceCount ,P ,T_Info ,Pnum ,d_BuildShell_SendCount ,STEP );


d_BuildShell_SendCount_DeviceToHost(d_BuildShell_SendCount, h_BuildShell_SendCount, deviceCount);
Build_Shell_ReceiveBuffer_From_SendBuffer(ReceBuffer , SendBuffer , h_BuildShell_SendCount , deviceCount , STEP );


Build_Shell_ReceiveCount_From_SendCount( d_BuildShell_ReceCount, d_BuildShell_SendCount , deviceCount);
Build_Shell_ReceCount_DeviceToHost( h_BuildShell_ReceCount ,  d_BuildShell_ReceCount , deviceCount) ;
Build_Shell_Insert_ReceBuffer_Fellow_AtomInfoArray(d_atoms , ReceBuffer , h_BuildShell_ReceCount , h_ParticleCount , h_AddShell_ParticleCount , STEP , deviceCount );

GlobalCellTable_Set_Zero( GlobalCellTable , T_Info , deviceCount );
Mapping_to_CellTable( d_atoms , GlobalCellTable , T_Info , h_AddShell_ParticleCount , deviceCount);


// Get force 
d_U_Set_Zero( d_U , deviceCount);
Computation_Force_Potential(d_atoms , h_ParticleCount ,GlobalCellTable , T_Info , d_U , U_lock , deviceCount);


// update kernel2
Update_AtomsInfo_kernel_2( d_atoms , h_ParticleCount , dt , deviceCount);


total_U = Get_Total_Potential(h_U, d_U, deviceCount);

 // Get the Kinetic on GPUs
 d_Ek_Set_Zero(d_EK, deviceCount);
 Computation_Kinetic(d_atoms , h_ParticleCount, d_EK, Signal, deviceCount);
 total_Ek =  Get_Total_Kinetic(h_EK, d_EK, deviceCount);
 printf("Step=%dth :\t kinetic is %f \t Potential is %f \n", step ,total_Ek/N , 0.5 * total_U / N );



 if(step<=Ne)
 {
   Scale_Velocity(d_atoms , h_ParticleCount , total_Ek * 2 , deviceCount , N);
 }


  All_devices_Synchronize(deviceCount);

 }



  free(atoms);
  free(Xseg);
  free(Yseg);
  free(Zseg);
  free(subdomain);

  for(int i=0;i<Pnum;i++)
    free(atomsInfoArr[i]);
  free(atomsInfoArr);

  free(h_ParticleCount);
  free(h_AddShell_ParticleCount);
  free(SubDomainBoundarySize);

  for(int i=0;i<Pnum;i++)
  {
    hipSetDevice(i);
    hipFree(d_atoms[i]);
  }
  free(d_atoms);


  for(int i=0;i<Pnum;i++)
  {
    hipSetDevice(i);
    hipFree(SendBuffer[i]);
  }
  free(SendBuffer);
  
  for(int i=0;i<Pnum;i++)
  {
    hipSetDevice(i);
    hipFree(ReceBuffer[i]);
  } 
  free(ReceBuffer);

  for(int i=0;i<Pnum;i++)
  {
    hipSetDevice(i);
    hipFree(d_BuildShell_SendCount[i]);
  }
  free(d_BuildShell_SendCount);


  for(int i=0;i<Pnum;i++)
    free(h_BuildShell_SendCount[i]);
  free(h_BuildShell_SendCount);

  for(int i=0;i<Pnum;i++)
  {
    hipSetDevice(i);
    hipFree(d_BuildShell_ReceCount[i]);
  } 
  free(d_BuildShell_ReceCount);


  for(int i=0;i<Pnum;i++)
    free(h_BuildShell_ReceCount[i]);
  free(h_BuildShell_ReceCount);

  for(int i=0;i<Pnum;i++)
  {
    hipSetDevice(i);
    hipFree(GlobalCellTable[i]);
  }
  free(GlobalCellTable);


  for(int i=0;i<Pnum;i++)
  {
    hipSetDevice(i);
    hipFree(d_U[i]);
  }
  free(d_U);

  for(int i=0;i<Pnum;i++)
  {
    hipSetDevice(i);
    hipFree(U_lock[i]);
  }
  free(U_lock);

  for(int i=0;i<Pnum;i++)
     free(h_U[i]);
  free(h_U);


  for(int i=0;i<Pnum;i++)
  {
    hipSetDevice(i);
    hipFree(d_EK[i]);
  }
  free(d_EK);

  for(int i=0;i<Pnum;i++)
  {
    hipSetDevice(i);
    hipFree(Signal[i]);
  }
  free(Signal);


  for(int i=0;i<Pnum;i++)
    free(h_EK[i]);
  free(h_EK);


  for(int i=0;i<Pnum;i++)
  {
    hipSetDevice(i);
    hipFree(d_Adjust[i]);
  }
  free(d_Adjust);


  for(int i=0;i<Pnum;i++)
  {
    hipSetDevice(i);
    hipFree(d_LingerCount[i]);
  }
  free(d_LingerCount);

  for(int i=0;i<Pnum;i++)
  {
    free(h_LingerCount[i]);
  }
  free( h_LingerCount);


  for(int i=0;i<Pnum;i++)
  {
    hipSetDevice(i);
    hipFree(d_Escape_SendCount[i]);
  }
  free(d_Escape_SendCount);


  for(int i=0;i<Pnum;i++)
    free(h_Escape_SendCount[i]);

  free(h_Escape_SendCount);


  for(int i=0;i<Pnum;i++)
  {
    hipSetDevice(i);
    hipFree(d_Escape_ReceCount[i]);
  } 
  free(d_Escape_ReceCount);
  

  for(int i=0;i<Pnum;i++)
    free(h_Escape_ReceCount[i]);

  free(h_Escape_ReceCount);

}
